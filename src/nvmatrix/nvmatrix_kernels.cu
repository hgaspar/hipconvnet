#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvmatrix_kernels.cuh>

__global__ void kTile(hipLaunchParm lp,const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    const int numThreads = hipBlockDim_x * hipGridDim_x;
    //    const unsigned int numEls = tgtWidth * tgtHeight;
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}

__global__ void kDotProduct_r(hipLaunchParm lp, float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * hipBlockIdx_x + hipThreadIdx_x;
    shmem[hipThreadIdx_x] = 0;
    if (eidx < numCols) {
        for (; eidx < numElements; eidx += numCols) {
            shmem[hipThreadIdx_x] += a[eidx] * b[eidx];
        }
    }
    __syncthreads();
    if (hipThreadIdx_x < 256) {
        shmem[hipThreadIdx_x] += shmem[hipThreadIdx_x + 256];
    }
    __syncthreads();
    if (hipThreadIdx_x < 128) {
        shmem[hipThreadIdx_x] += shmem[hipThreadIdx_x + 128];
    }
    __syncthreads();
    if (hipThreadIdx_x < 64) {
        shmem[hipThreadIdx_x] += shmem[hipThreadIdx_x + 64];
    }
    __syncthreads();
    if (hipThreadIdx_x < 32) {
        volatile float* mysh = &shmem[hipThreadIdx_x];
        *mysh += mysh[32];
        *mysh += mysh[16];
        *mysh += mysh[8];
        *mysh += mysh[4];
        *mysh += mysh[2];
        *mysh += mysh[1];
        if (hipThreadIdx_x == 0) {
            target[hipBlockIdx_x] = *mysh;
        }
    }
}

__global__ void kSetupCurand(hipLaunchParm lp, hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * hipBlockIdx_x + hipThreadIdx_x;
    /* Each thread gets same seed, a different sequence number,
     no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}

